#include "hip/hip_runtime.h"
/*
 *  (C) copyright  2011, Ismael Garcia, (U.Girona/ViRVIG, Spain & INRIA/ALICE, France)
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#ifndef EXAMPLE_01_CU_
#define EXAMPLE_01_CU_

//------------------------------------------------------------------------

#define ENABLE_KEY_DATA_MODE                 1 // Has 64b (key+data) / 32b (key only)
#define ENABLE_2D_COORDS                     1 // Pack 2D coords as keys
#define ENABLE_3D_COORDS                     0 // Pack 3D coords as keys
#define ENABLE_LIBHU_GPU_RANDOM_GENERATOR    0 // Use faster random values generator
#define ENABLE_LIBHU_LOG                     1 // Show log of libhu functions
#define ENABLE_KERNEL_PROFILING              0 // Enable kernel profiling

//------------------------------------------------------------------------

#include <iostream>
#include <iomanip>

#include <libh/hash.h>
#include <libhu/hash_utils.h>

#include <key_coh_hash.h>
#include <key_rand_hash.h>
#include <key_value_coh_hash.h>
#include <key_value_rand_hash.h>

#include <config_params.h>
#include <img_tga.inl>
#include <mt19937ar.h>

#include <test_cu_robin_hood_hash.cu>

//------------------------------------------------------------------------

void runRobinHoodTest(ConfigParams& cfg) 
{
  testRobinHoodHash(cfg);
}

//------------------------------------------------------------------------

int runHashTest(int argc, char** argv, ConfigParams& cfg)
{
  if (argc < 6)
  {
    std::cerr << "* Random numbers hashing:"                                                                    << std::endl;
    std::cerr << "phash.exe [num_keys] [access_rate_non_valid_keys] [density] [seed] [access_mode]"             << std::endl;
    std::cerr << "  [num_keys]                   -- Integer number of keys"                                     << std::endl;
    std::cerr << "  [access_rate_non_valid_keys] -- Integer percentage 0-100"                                   << std::endl;
    std::cerr << "                                  (e.g. 20, means a total query of 20% non-valid"             << std::endl;
    std::cerr << "                                  accessed keys + [num_keys] of 80% valid keys)"              << std::endl;
    std::cerr << "  [density]                    -- Integer percentage 0-100"                                   << std::endl;
    std::cerr << "  [seed]                       -- Integer seed"                                               << std::endl;
    std::cerr << "  [access_mode]                -- 1 (sorted) / 0 (random shuffle)"                            << std::endl;
    std::cerr << std::endl;
    std::cerr << "(e.g. 'example_01.exe 32000000 100 80 61332125 1 -coh_hash')"                                 << std::endl;
    std::cerr << std::endl;
    std::cerr << "* Image data hashing:"                                                                        << std::endl;
    std::cerr << "phash.exe [image_file] [access_null_keys] [density] [seed] [access_mode]"                     << std::endl;
    std::cerr << "  [image_file]                 -- image data"                                                 << std::endl;
    std::cerr << "                                  (Use uncompressed tga images with RGBA"                     << std::endl;
    std::cerr << "                                   channels, value '#00000000' identify"                      << std::endl;
    std::cerr << "                                   non-valid key-data pixel entries)"                         << std::endl;
    std::cerr << "  [access_null_keys]           -- 1 access valid & non-valid keys /"                          << std::endl;
    std::cerr << "                                  0 access only valid keys"                                   << std::endl;
    std::cerr << "  [density]                    -- Integer percentage 0-100"                                   << std::endl;
    std::cerr << "  [seed]                       -- Integer seed"                                               << std::endl;
    std::cerr << "  [access_mode]                -- 1 (sorted) / 0 (random shuffle)"                            << std::endl;
    std::cerr << std::endl;
    std::cerr << "(e.g. 'example_01.exe flower_1024.tga 1 80 77016577 1 -coh_hash')"                            << std::endl;
    std::cerr << std::endl;
    
    return 0;
  }

  std::vector<std::string> sparams(argv, argv+argc);
  
  size_t found;
  found=sparams[1].find(".tga");
  if (found!=std::string::npos)
  {
    std::cerr << "image_data mode enabled" << std::endl;

    cfg.rand_num_mode     = false;
    cfg.image_mode        = true;
    cfg.image_name        = sparams[1];
    cfg.access_null_keys  = bool(atoi(sparams[2].c_str()));
    cfg.dens              = libhu::F32(atoi(sparams[3].c_str())) / 100.0f;
    cfg.seed              = atoi(sparams[4].c_str());
    cfg.sorted_access     = atoi(sparams[5].c_str());

    cfg.tex = loadTGA((char*)cfg.image_name.c_str());

    saveTGA(cfg.tex, "image_to_hash.tga");

    libhu::U32 tnnz = 0;
    for (libhu::U32 i = 0; i < cfg.tex->w * cfg.tex->h; i++)
    {
      libhu::U32 *imgPtr = (libhu::U32*)cfg.tex->data;
      if (imgPtr[i] != 0)
      {
        tnnz++;
      }
    }
    cfg.num_keys = tnnz;
    cfg.num_extra = cfg.tex->w * cfg.tex->h;
    if (cfg.access_null_keys)
    {
      cfg.rate_non_valid_keys = (float)(cfg.num_extra - cfg.num_keys) / (float)cfg.num_extra;
    }
    else
    {
      cfg.rate_non_valid_keys = 0.0;
      cfg.num_extra      = cfg.num_keys;
    }

  }
  else
  {
    cfg.rand_num_mode       = true;
    cfg.image_mode          = false;
    cfg.num_keys            = atoi(sparams[1].c_str());
    cfg.rate_non_valid_keys = (atoi(sparams[2].c_str()) / 100.0f);
    cfg.num_extra           = (cfg.num_keys / (1.0 - (atoi(sparams[2].c_str()) / 100.0f)));
    cfg.dens                = libhu::F32(atoi(sparams[3].c_str())) / 100.0f;
    cfg.seed                = atoi(sparams[4].c_str());
    cfg.sorted_access       = atoi(sparams[5].c_str());
  }
  
  if (argc > 6)
  {
    cfg.coh_hash          = (sparams[6] == "-coh_hash") ? 1 : 0;
    cfg.rand_hash         = (sparams[6] == "-rand_hash") ? 1 : 0;
  }
  else
  {
    cfg.coh_hash          = 1;
    cfg.rand_hash         = 1;
  }

  cfg.is_set = false;

  // Default 2D universe size
  cfg.u2D_w = 16384;
  cfg.u2D_h = 16384;

  // Default 3D universe size
  cfg.u3D_w = 512;
  cfg.u3D_h = 512;
  cfg.u3D_d = 512;

  runRobinHoodTest(cfg);

  libhu::F32 TIME_1K_MILLISECONDS = 1000;
  libhu::F32 NUM_1M_KEYS = 1000000;
  libhu::F32 build_keys  = cfg.num_keys / NUM_1M_KEYS;
  libhu::F32 access_keys = ((cfg.rate_non_valid_keys == 0) ? cfg.num_keys : cfg.num_extra) / NUM_1M_KEYS;

  if (cfg.coh_hash)
  {
    std::cerr << "rh_coh_hash                      : " << cfg.rh_coh_hash_state << std::endl;
    std::cerr << "build rh_coh_hash                : " << std::setiosflags(std::ios::fixed) << std::setprecision(4) << cfg.rh_coh_hash_build_keys_per_sec << " Mkeys/sec" << std::endl;
    std::cerr << "access rh_coh_hash               : " << std::setiosflags(std::ios::fixed) << std::setprecision(4) << cfg.rh_coh_hash_access_keys_per_sec << " Mkeys/sec" << std::endl;
    std::cerr << std::endl;
    std::cerr << "-------------------------------------------" << std::endl;
  }
  else if (cfg.rand_hash)
  {
    std::cerr << "rh_rand_hash                     : " << cfg.rh_rand_hash_state << std::endl;
    std::cerr << "build rh_rand_hash               : " << std::setiosflags(std::ios::fixed) << std::setprecision(4) << cfg.rh_rand_hash_build_keys_per_sec << " Mkeys/sec" << std::endl;
    std::cerr << "access rh_rand_hash              : " << std::setiosflags(std::ios::fixed) << std::setprecision(4) << cfg.rh_rand_hash_access_keys_per_sec << " Mkeys/sec" << std::endl;
    std::cerr << std::endl;
    std::cerr << "-------------------------------------------" << std::endl;
  }

  if (cfg.image_mode)
  {
    delete cfg.tex;
  }

}

//------------------------------------------------------------------------

int main(int argc,char **argv)
{

  ConfigParams cfg;
  runHashTest(argc, argv, cfg);
  
  return 0;

}


#endif